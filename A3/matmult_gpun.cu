#include "matmult_kernels.h"

extern "C"{

    void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C){
        //allocate memory on GPU
        double* d_A;
        double* d_B;
        double* d_C;
        hipMalloc((void**) &d_A, m*k*sizeof(double));
        hipMalloc((void**) &d_B, n*k*sizeof(double));
        hipMalloc((void**) &d_C, m*n*sizeof(double));

        //move A and B to GPU
        hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, n*k*sizeof(double), hipMemcpyHostToDevice);
        
        //call kernel
        matmult_kernel1<<<1,1>>>(m, n, k, A, B, C);
        hipDeviceSynchronize();

        //move C back to host
        hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);
    }

    void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C){
        
    }

    void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C){
        
    }

    void matmult_gpu4(int m, int n, int k, double *A, double *B, double *C){
        
    }

}