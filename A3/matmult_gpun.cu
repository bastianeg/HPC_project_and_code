#include "hip/hip_runtime.h"
#include "matmult_kernels.h"
#include <stdio.h>
#include "matmult_gpu5.h"
extern "C"{

    void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C){
        
        //allocate memory on GPU
        double* d_A;
        double* d_B;
        double* d_C;
        hipMalloc((void**) &d_A, m*k*sizeof(double));
        hipMalloc((void**) &d_B, n*k*sizeof(double));
        hipMalloc((void**) &d_C, m*n*sizeof(double));

        //move A and B to GPU
        hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, n*k*sizeof(double), hipMemcpyHostToDevice);

        //call kernel
        matmult_kernel1<<<1,1>>>(m, n, k, d_A, d_B, d_C);
        hipDeviceSynchronize();

        //move C back to host
        hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

    }

    void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C){
        //allocate memory on GPU
        double* d_A;
        double* d_B;
        double* d_C;
        hipMalloc((void**) &d_A, m*k*sizeof(double));
        hipMalloc((void**) &d_B, n*k*sizeof(double));
        hipMalloc((void**) &d_C, m*n*sizeof(double));

        //move A and B to GPU
        hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, n*k*sizeof(double), hipMemcpyHostToDevice);

        //number of blocks is ceil of N/bs 
        int bs = 32;
        int mblocks = m/bs + (int) (m%bs!=0);
        int nblocks = n/bs + (int) (n%bs!=0);

        //call kernel
        matmult_kernel2<<<dim3 (mblocks,nblocks),dim3 (bs,bs)>>>(m, n, k, d_A, d_B, d_C);
        hipDeviceSynchronize();

        //move C back to host
        hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C){
        //allocate memory on GPU
        double* d_A;
        double* d_B;
        double* d_C;
        hipMalloc((void**) &d_A, m*k*sizeof(double));
        hipMalloc((void**) &d_B, n*k*sizeof(double));
        hipMalloc((void**) &d_C, m*n*sizeof(double));

        //move A and B to GPU
        hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, n*k*sizeof(double), hipMemcpyHostToDevice);

        //number of blocks is ceil of N/bs 
        int bs = 32;
        int mblocks = m/bs + (int) (m%bs!=0);
        int nblocks = n/bs/2 + (int) (n%(bs*2)!=0);

        //call kernel
        matmult_kernel3<<<dim3 (mblocks,nblocks),dim3 (bs,bs)>>>(m, n, k, d_A, d_B, d_C);
        hipDeviceSynchronize();

        //move C back to host
        hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    void matmult_gpu4(int m, int n, int k, double *A, double *B, double *C){
        //number of elements to compute in each thread
        int s = atoi(getenv("NUM_ELEM_PER_THREAD"));
        //allocate memory on GPU
        double* d_A;
        double* d_B;
        double* d_C;
        hipMalloc((void**) &d_A, m*k*sizeof(double));
        hipMalloc((void**) &d_B, n*k*sizeof(double));
        hipMalloc((void**) &d_C, m*n*sizeof(double));

        //move A and B to GPU
        hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, n*k*sizeof(double), hipMemcpyHostToDevice);

        //number of blocks is ceil of N/bs 
        int bs = 32;
        int mblocks = m/bs + (int) (m%bs!=0);
        int nblocks = n/bs/s + (int) (n%(bs*s)!=0);

        //call kernel
        matmult_kernel4<<<dim3 (mblocks,nblocks),dim3 (bs,bs)>>>(m, n, k, d_A, d_B, d_C, s);
        hipDeviceSynchronize();

        //move C back to host
        hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    void matmult_gpu5(int m, int n, int k, double *A, double *B, double *C){
        // Load A and B to device memory
        double* d_A;
        double* d_B;
        double* d_C;

        hipMalloc((void **)&d_A,  m * k * sizeof(double));
        hipMalloc((void **)&d_B,  k * n * sizeof(double));
        hipMalloc((void **)&d_C,  n * m * sizeof(double));

        hipMemcpy(d_A, A, m * k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k * n * sizeof(double), hipMemcpyHostToDevice);

        #define BLOCK_SIZE 16
        // Invoke kernel
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(n / dimBlock.x, m / dimBlock.y);
        MatMulKernel5<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

        // Read C from device memory
        hipMemcpy(C, d_C, n * m * sizeof(double), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        
    }
}