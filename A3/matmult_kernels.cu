#include "hip/hip_runtime.h"

__global__ void
matmult_kernel1(int m, int n, int k, double *A, double *B, double *C){
    // set C to zeros
    for (int i=0;i<m;i++){
        for (int p=0;p<n;p++){
            C[i][p]=0;
        }
    }
    // do matmult with mkn loop order
    for (int i=0;i<m;i++) {
        for (int j=0;j<k;j++){
            for (int p=0;p<n;p++) { 
                C[i][p]+=A[i][j]*B[j][p];
            }
        }
    }
}