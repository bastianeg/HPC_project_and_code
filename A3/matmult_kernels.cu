
#include <hip/hip_runtime.h>
#include <stdio.h>


#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__global__ void
matmult_kernel1(int m, int n, int k, double *A, double *B, double *C){
    
    // set C to zeros
    for (int i=0;i<m;i++){
        for (int p=0;p<n;p++){
            C[i*n+p]=0; //C[i][p]
        }
    }
    // do matmult with mkn loop order
    for (int i=0;i<m;i++) {
        for (int j=0;j<k;j++){
            for (int p=0;p<n;p++) { 
                C[i*n+p] += A[i*k+j] * B[j*n+p];
            }
        }
    }
}

__global__ void
matmult_kernel2(int m, int n, int k, double *A, double *B, double *C){
    
    int i = blockIdx.x*blockDim.x+threadIdx.x; //looping through m
    int j = blockIdx.y*blockDim.y+threadIdx.y; //looping through n

    if((i<m)&&(j<n)){
        //init C to zero
        C[i*n+j] = 0.0;
        for(int p=0; p<k; p++){
            //read row of A and col of B
            C[i*n+j] += A[i*k+p] * B[p*n+j];
        }
    }
}

__global__ void
matmult_kernel3(int m, int n, int k, double *A, double *B, double *C){
    //compute C(i,j) and C(i,j+1)
    int i = blockIdx.x*blockDim.x+threadIdx.x; //looping through m
    int j = 2*(blockIdx.y*blockDim.y+threadIdx.y); //looping through n (only half as many threads/blocks)
    
    

    if((i<m)&&(j<n)){
        //additional j to compute (here, either 1 or 0)
        int j_add = MIN(1,n-1-j);
        
        //init C to zero
        #pragma unroll
        for(int u=0; u<=j_add; u++){
            C[i*n+j+u] = 0.0;
        }
        C[i*n+j+1] = 0.0;
        for(int p=0; p<k; p++){
            //row of A and col of B
            #pragma unroll
            for(int u=0; u<=j_add; u++){
                C[i*n+j+u] += A[i*k+p] * B[p*n+j+u];
            }
        }
    }
}

__global__ void
matmult_kernel4(int m, int n, int k, double *A, double *B, double *C,int s){
    //compute C(i,j), C(i,j+1), ... C(i,j+s)
    int i = blockIdx.x*blockDim.x+threadIdx.x; //looping through m
    int j = s*(blockIdx.y*blockDim.y+threadIdx.y); //looping through n (only 1/s as many threads/blocks)

    if((i<m)&&(j<n)){
        //additional j to compute (here, from 0 to s-1)
        int j_add = MIN(s-1,n-1-j);
        
        //init C to zero
        #pragma unroll
        for(int u=0; u<=j_add; u++){
            C[i*n+j+u] = 0.0;
        }
        C[i*n+j+1] = 0.0;
        
        for(int p=0; p<k; p++){
            //row of A and col of B
            #pragma unroll
            for(int u=0; u<=j_add; u++){
                C[i*n+j+u] += A[i*k+p] * B[p*n+j+u];
            }
        }
    }
}




/*
BEGIN GPU 5
##############################################################################
*/

#define BLOCK_SIZE 16

typedef struct {
    int width;
    int height;
    int stride; 
    double* elements;
} Matrix;


// Get a matrix element
__device__ double GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           double value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col) 
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return Asub;
}

// Matrix multiplication kernel called by MatMul()
__global__ void gpu5_kernel(const Matrix A, const Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    double Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}