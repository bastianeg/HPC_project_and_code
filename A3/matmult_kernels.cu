
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void
matmult_kernel1(int m, int n, int k, double *A, double *B, double *C){
    
    
    for(int i = 0; i<k; i++){
            for(int j = 0; j<n; j++){
                printf("%.2lf ",B[i*n+j]);
            }
            printf("\n");
        }
        printf("is\n");
    
    // set C to zeros
    for (int i=0;i<m;i++){
        for (int p=0;p<n;p++){
            C[i*n+p]=0; //C[i][p]
        }
    }
    // do matmult with mkn loop order
    for (int i=0;i<m;i++) {
        for (int j=0;j<k;j++){
            for (int p=0;p<n;p++) { 
                C[i*n+p] += A[i*k+j] * B[j*n+p];
            }
        }
    }

    for(int i = 0; i<m; i++){
            for(int j = 0; j<n; j++){
                printf("%.2lf ",C[i*n+j]);
            }
            printf("\n");
        }

}