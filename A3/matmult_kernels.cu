
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void
matmult_kernel1(int m, int n, int k, double *A, double *B, double *C){
    
    // set C to zeros
    for (int i=0;i<m;i++){
        for (int p=0;p<n;p++){
            C[i*n+p]=0; //C[i][p]
        }
    }
    // do matmult with mkn loop order
    for (int i=0;i<m;i++) {
        for (int j=0;j<k;j++){
            for (int p=0;p<n;p++) { 
                C[i*n+p] += A[i*k+j] * B[j*n+p];
            }
        }
    }
}

__global__ void
matmult_kernel2(int m, int n, int k, double *A, double *B, double *C){
    
    int i = blockIdx.x*blockDim.x+threadIdx.x; //looping through m
    int j = blockIdx.y*blockDim.y+threadIdx.y; //looping through n

    if((i<m)&&(j<n)){
        //init C to zero
        C[i*n+j] = 0.0;
        for(int p=0; p<k; p++){
            //read row of A and col of B
            C[i*n+j] += A[i*k+p] * B[p*n+j];
        }
    }
}

__global__ void
matmult_kernel3(int m, int n, int k, double *A, double *B, double *C){
    //compute C(i,j) and C(i,j+1)
    int i = blockIdx.x*blockDim.x+threadIdx.x; //looping through m
    int j = 2*(blockIdx.y*blockDim.y+threadIdx.y); //looping through n (only half as many threads/blocks)

    if((i<m)&&(j<n)){
        //init C to zero
        C[i*n+j] = 0.0;
        C[i*n+j+1] = 0.0;
        for(int p=0; p<k; p++){
            //read row of A and col of B
            C[i*n+j] += A[i*k+p] * B[p*n+j];
            C[i*n+j+1] += A[i*k+p] * B[p*n+j+1];
        }
    }
}